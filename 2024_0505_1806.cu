#include "hip/hip_runtime.h"
#include<stdio.h>
#include<time.h>
#include<stdlib.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define BS 1024
#define WALL_GPU  (1024*1024-1)
#define ROAD 0 
#define WIDTH 128 
#define HEIGHT 64 
#define MAPSIZE (WIDTH*HEIGHT) 
#define WALL (MAPSIZE*8 -1)
//#define N  524288/* 配列の長さ、2の30乗 
 /* GPUカーネル関数の定義*/
 __global__ void bidirectional_search(int *kk,int *Dend,int *Barray,int *j,int *DA,int *s,int *g,int *start,int *goal,int *end_sg)
 {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i>=(((MAPSIZE)*(MAPSIZE)))){
	return;
	}
    __syncthreads();
	if(*end_sg==1){	
			return;
	}
		
		
    __syncthreads();
		if(*j==1){
    __syncthreads();
			Barray[i]=0;
    __syncthreads();
		if(MAPSIZE>i){
			Dend[i]=WALL;
    __syncthreads();
			s[i]=0;
    __syncthreads();
			kk[i]=0;
    __syncthreads();
			g[i]=0;
		}
		}
	s[*start]=1;
    __syncthreads();
	kk[*start]=((MAPSIZE)/2);
    __syncthreads();
	Barray[(*start)*MAPSIZE+(i%MAPSIZE)]=1;
    __syncthreads();
	g[*goal]=1;
    __syncthreads();
	kk[*goal]=0;
    __syncthreads();
	Barray[(*goal)*MAPSIZE+(i%MAPSIZE)]=1;
    __syncthreads();
	if((i/MAPSIZE)!=(*start)&&((i/MAPSIZE)!=(*goal))&&DA[i/MAPSIZE]!=WALL&&DA[i%MAPSIZE]!=WALL){
    __syncthreads();
			if(((s[(i)/MAPSIZE+1]|s[(i)/MAPSIZE-1]|s[(i)/MAPSIZE+WIDTH]|s[(i)/MAPSIZE-WIDTH])>=1)&&((g[(i)/MAPSIZE+1]|g[(i)/MAPSIZE-1]|g[(i)/MAPSIZE+WIDTH]|g[(i)/MAPSIZE-WIDTH])>=1)&&s[i/MAPSIZE]==0&&g[i/MAPSIZE]==0){
    __syncthreads();
				kk[i/MAPSIZE]=((MAPSIZE)/2)-(*j);
    __syncthreads();
				*end_sg=1;	
    __syncthreads();
					Barray[i/MAPSIZE*MAPSIZE+i%MAPSIZE]=Barray[(i/MAPSIZE+WIDTH)*MAPSIZE+i%MAPSIZE]|Barray[(i/MAPSIZE-1)*MAPSIZE+i%MAPSIZE]|Barray[(i/MAPSIZE+1)*MAPSIZE+i%MAPSIZE]|Barray[i/MAPSIZE-WIDTH*MAPSIZE+i%MAPSIZE];
    
    __syncthreads();
			if(Barray[i/MAPSIZE*MAPSIZE+i%MAPSIZE]==1){
    __syncthreads();
					Dend[i%MAPSIZE]=kk[i%MAPSIZE];
    __syncthreads();
			}
			s[i/MAPSIZE]=s[(i)/MAPSIZE+1]|s[i/MAPSIZE-1]|s[(i)/MAPSIZE+WIDTH]|s[(i)/MAPSIZE-WIDTH];
    __syncthreads();
			g[i/MAPSIZE]=g[(i)/MAPSIZE+1]|g[i/MAPSIZE-1]|g[(i)/MAPSIZE+WIDTH]|g[(i)/MAPSIZE-WIDTH];
    __syncthreads();
return;
}
		if(((s[((i)/MAPSIZE)]|s[(i)/MAPSIZE-1]|s[(i)/MAPSIZE+WIDTH]|s[(i)/MAPSIZE-WIDTH])>=1||(g[(i)/MAPSIZE+1]|g[(i)/MAPSIZE-1]|g[(i)/MAPSIZE+WIDTH]|g[(i)/MAPSIZE-WIDTH])>=1)){
    __syncthreads();

		 if(s[i/MAPSIZE]==0&&(s[i/MAPSIZE+1]|s[i/MAPSIZE-1]|s[i/MAPSIZE+WIDTH]|s[i/MAPSIZE-WIDTH])==1){
    __syncthreads();
				kk[i/MAPSIZE]=((MAPSIZE)/2)-(*j);
    __syncthreads();
					Barray[i/MAPSIZE*MAPSIZE+i%MAPSIZE]=Barray[(i/MAPSIZE+WIDTH)*MAPSIZE+i%MAPSIZE]|Barray[(i/MAPSIZE-1)*MAPSIZE+i%MAPSIZE]|Barray[(i/MAPSIZE+1)*MAPSIZE+i%MAPSIZE]|Barray[i/MAPSIZE-WIDTH*MAPSIZE+i%MAPSIZE];
    __syncthreads();
    				kk[i/MAPSIZE]=((MAPSIZE)/2)-(*j);
    __syncthreads();
return;
		}
		 if(g[i/MAPSIZE]==0&&(g[i/MAPSIZE+1]|g[i/MAPSIZE-1]|g[i/MAPSIZE+WIDTH]|g[i/MAPSIZE-WIDTH])==1){
    __syncthreads();
				kk[i/MAPSIZE]=(*j);
    __syncthreads();
					Barray[i/MAPSIZE*MAPSIZE+i%MAPSIZE]=Barray[(i/MAPSIZE+WIDTH)*MAPSIZE+i%MAPSIZE]|Barray[(i/MAPSIZE-1)*MAPSIZE+i%MAPSIZE]|Barray[(i/MAPSIZE+1)*MAPSIZE+i%MAPSIZE]|Barray[i/MAPSIZE-WIDTH*MAPSIZE+i%MAPSIZE];
    __syncthreads();
        __syncthreads();
			g[i/MAPSIZE]=g[(i)/MAPSIZE+1]|g[i/MAPSIZE-1]|g[(i)/MAPSIZE+WIDTH]|g[(i)/MAPSIZE-WIDTH];
    __syncthreads();
return;
		}
return;
}
return;
}
return;
}
int main(void)
 {
 int *i;
// int j;
 int *start_gpu,*goal_gpu;	
 int *start,*goal;	
 int *start_flag,*goal_flag;
 int *Dstart_flag,*Dgoal_flag;
 int *ROUTE;
 int *A; /* ホストメモリ用のポインタ*/
 long start_time,end_time,pre_time_start,pre_time_end;
 int *ii;
 int *end,*gend;
 int *array;
 int *wall_end;
 int *Dwall_end;
 int *Darray;
 int j=0;
 int *kkcpu;
 int *kkgpu;
	printf("\nney\n");
  kkcpu=(int *)malloc(sizeof(int)*(MAPSIZE)); /* 配列Aの領域確保*/
 wall_end = (int *)malloc((MAPSIZE) *sizeof(int));
 array = (int *)malloc((MAPSIZE)*(MAPSIZE)*sizeof(int));
     // 各行ごとに列数分のメモリを確保
  ROUTE=(int *)malloc(sizeof(int)*(MAPSIZE)); /* 配列Aの領域確保*/
  goal_flag = (int *)malloc(sizeof(int)*(MAPSIZE)); /* 配列Aの領域確保*/
  start = (int *)malloc(sizeof(int)); /* 配列Aの領域確保*/
  goal = (int *)malloc(sizeof(int)); /* 配列Aの領域確保*/
  i = (int *)malloc(sizeof(int)); /* 配列Aの領域確保*/
		printf("\nney\n");
  start_flag = (int *)malloc(sizeof(int)*(MAPSIZE)); /* 配列Aの領域確保*/
		printf("\nney\n");
		printf("\nney\n");
		printf("\nney\n");
  end = (int *)malloc(sizeof(int)); /* 配列Aの領域確保*/
		printf("\nney\n");
  *end=0;
		printf("\nney\n");
  *start=WIDTH+1;
		printf("\nney\n");
  *goal=(MAPSIZE)-WIDTH-2;
		printf("\nney\n");
  start_flag[*start]=1;
		printf("\nney\n");
  goal_flag[*goal]=1;
		printf("\nneyggggggggg\n");
  	  for(int in=0;in<(MAPSIZE);in++){
		if((in/WIDTH)==0||(in/WIDTH)==HEIGHT-1){
				ROUTE[in]=WALL;
		}else if((in%WIDTH)==0||(in%WIDTH)==WIDTH-1){
				ROUTE[in]=WALL;
		}else if((in/WIDTH)%2==1){
				ROUTE[in]=ROAD;
		}else if((in/WIDTH)%2==0){
			if((in%WIDTH)%2==0){
				ROUTE[in]=ROAD;
			}else{
				ROUTE[in]=WALL;
			}
		}
		wall_end[*i]=0;
	}
 /* 配列A (GPU) の領域確保*/
 pre_time_start=clock();
 hipMalloc((int**)&kkgpu, sizeof(int)*(MAPSIZE)); 
 hipMalloc((int**)&Dwall_end, sizeof(int)*MAPSIZE); 
 hipMalloc((int**)&Darray, sizeof(int)*(MAPSIZE)*(MAPSIZE)); 
	printf("s%04d,g%d\n",*start,*goal);
 hipMalloc((int**)&ii, sizeof(int)); 
 hipMalloc((int**)&A, sizeof(int)*(MAPSIZE)); 
 hipMalloc((int**)&Dgoal_flag, sizeof(int)*(MAPSIZE)); 
 hipMalloc((int**)&Dstart_flag, sizeof(int)*(MAPSIZE)); 
 hipMalloc((int**)&start_gpu, sizeof(int)); 
 hipMalloc((int**)&goal_gpu, sizeof(int)); 
 hipMalloc((int**)&gend, sizeof(int)); 
/* ホストメモリからデバイスメモリへコピー*/
 hipMemcpy( start_gpu,start, sizeof(int), hipMemcpyDefault);
 hipMemcpy( goal_gpu,goal, sizeof(int), hipMemcpyDefault);
 pre_time_end=clock();
	printf("pre_time=%ld\n",pre_time_end-pre_time_start);
	for(*i=0;*i<(MAPSIZE);*i=(*i+1)){	
		printf("%4d,",ROUTE[*i]);
		if((*i%WIDTH)==(WIDTH-1)){
			printf("\n");
		}
	}
 /* GPUカーネル関数呼び出し*/
	start_time=clock();
	for(*i=1;(*i)<MAPSIZE;*i=(*i+1)){
 		hipMemcpy( kkgpu,kkcpu, sizeof(int)*(MAPSIZE), hipMemcpyDefault);
 		hipMemcpy( Dwall_end,wall_end, sizeof(int)*(MAPSIZE), hipMemcpyDefault);
	 	hipMemcpy(Darray ,array,sizeof(int)*(MAPSIZE)*(MAPSIZE),hipMemcpyDefault); 


 		hipMemcpy( ii,i, sizeof(int), hipMemcpyDefault);
 		hipMemcpy( A,ROUTE, sizeof(int)*(MAPSIZE), hipMemcpyDefault);
 		hipMemcpy( Dgoal_flag,goal_flag, sizeof(int)*(MAPSIZE), hipMemcpyDefault);
 		hipMemcpy( Dstart_flag,start_flag, sizeof(int)*(MAPSIZE), hipMemcpyDefault);
 		hipMemcpy( gend,end ,sizeof(int), hipMemcpyDefault);
		bidirectional_search<<<((MAPSIZE*MAPSIZE)+BS-1)/BS, BS,2>>>(kkgpu,Dwall_end ,Darray   ,ii,A,Dstart_flag,Dgoal_flag,start_gpu,goal_gpu,gend);
		hipDeviceSynchronize();
 		hipMemcpy( wall_end,Dwall_end, sizeof(int)*(MAPSIZE), hipMemcpyDefault);
		hipMemcpy(array,Darray ,sizeof(int)*(MAPSIZE)*(MAPSIZE),hipMemcpyDefault); 
 		hipMemcpy( ROUTE,A, sizeof(int)*(MAPSIZE), hipMemcpyDefault);
 		hipMemcpy( start_flag,Dstart_flag, sizeof(int)*(MAPSIZE), hipMemcpyDefault);
 		hipMemcpy( goal_flag,Dgoal_flag, sizeof(int)*(MAPSIZE), hipMemcpyDefault);
 		hipMemcpy( end,gend ,sizeof(int), hipMemcpyDefault);
 		hipMemcpy( kkcpu,kkgpu, sizeof(int)*(MAPSIZE), hipMemcpyDefault);
		if(*end==1){
				j++;
				if(j>1){
					break;
				}
			
}
}
	hipFree(Dstart_flag);
	end_time=clock();
	for(*i=0;*i<(MAPSIZE);*i=(*i+1)){	
		printf("%4d,",wall_end[*i]);
		ROUTE[*i]=wall_end[*i];
		if((*i%WIDTH)==(WIDTH-1)){
			printf("\n");
		}
	}
	hipFree(Dstart_flag);
	hipFree(Dgoal_flag);
	hipFree(A);
	hipFree(A);
	hipFree(Dwall_end);
	hipFree(start_gpu);
	hipFree(ii);
	hipFree(goal_gpu);
	hipFree(Darray);
	hipFree(gend);
	hipFree(Dwall_end);
	free(i);
	free(end);
	free(array);
	free(wall_end);
	printf("s%04d,g%d\n",*start,*goal);
	printf("exe_time=%ld\n,",(end_time-start_time));
		int nowplot,nextplot;
		int count;
		int *result;
		  result= (int *)malloc(sizeof(int)*(MAPSIZE)); /* 配列Aの領域確保*/
					nowplot=(*start);
					nextplot=(*start);
						
					count=0;
					while(nowplot!=(*goal)){
						printf("%d ", nowplot);
						result[count]=nowplot;
						if(ROUTE[nowplot-1]!=-1){
							if(ROUTE[nextplot]>=ROUTE[nowplot-1]){
								nextplot=nowplot-1;
							}
						}
						if(ROUTE[nowplot+1]!=-1){
							if(ROUTE[nextplot]>=ROUTE[nowplot+1]){
								nextplot=nowplot+1;
							}
						}
						if(ROUTE[nowplot+WIDTH]!=-1){
							if(ROUTE[nextplot]>=ROUTE[nowplot+WIDTH]){
								nextplot=nowplot+WIDTH;
							}
						}
						if(ROUTE[nowplot-WIDTH]!=-1){
							if(ROUTE[nextplot]>=ROUTE[nowplot-WIDTH]){
								nextplot=nowplot-WIDTH;
							}
						}
						nowplot=nextplot;        
						count++;
					}
					result[count]=(*goal);
					count=0;
					while(result[count]!=(*goal)){
						printf(" %d ",result[count]);
						count++;
					}
	free(ROUTE);
	free(start);
	free(goal);
	printf("exe_time=%ld\n,",(end_time-start_time));
	return 0;
 }
