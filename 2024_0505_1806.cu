#include "hip/hip_runtime.h"
#include<stdio.h>
#include<time.h>
#include<stdlib.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define BS 1024
#define WALL 1023 
#define WALL_GPU  (1024*1024-1)
#define ROAD 0 
#define WIDTH 16 
#define HEIGHT 8 
#define MAPSIZE ((int)WIDTH*HEIGHT) 
//#define N  524288/* 配列の長さ、2の30乗 
#define BS 1024 
 /* GPUカーネル関数の定義*/
 __global__ void bidirectional_search(int *j,int *DA,int *s,int *g,int *start,int *goal,int *count,int *end_sg)
 {
	int loop=0;
	int  sync_num=0;
	int ste[MAPSIZE];
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int map_in[MAPSIZE];
//	int s[MAPSIZE],g[MAPSIZE];
	if(i>=MAPSIZE){

		return;
	}
			
	if(*j%2==0){
		if((i/WIDTH)%2==0){
			if(i%2==0){
				return;	
			}
		}else{
			if(i%2==1){
				return;	
			}
		}
	}else{
		if((i/WIDTH)%2==1){
			if(i%2==0){
				return;	
			}
		}else{
			if(i%2==1){
				return;	
			}
		}
	}
	
//	A[i]=1;
//		return;
		//}

	if(DA[i]!=WALL){
		if(!(s[i]>=1&&g[i]>=1)){
		if(s[i]!=(s[i+1]|s[i-1]|s[i+WIDTH]|s[i-WIDTH])){
			DA[i]=(MAPSIZE/2)-*j;
		}
		if(g[i]!=(g[i+1]|g[i-1]|g[i+WIDTH]|g[i-WIDTH])){
			DA[i]=*j;
		}
		}
//	for(loop=1;loop<(MAPSIZE);++loop){
//		[	if(i!=1){	ZZ++
			s[i]=s[i+1]|s[i-1]|s[i+WIDTH]|s[i-WIDTH];
			g[i]=g[i+1]|g[i-1]|g[i+WIDTH]|g[i-WIDTH];
//			}
			if(s[i]>=1&&g[i]>=1){
				DA[i]=*j;
				*end_sg=1;	
			}
								

//		}
//	}	}
	 
		//	break;
		//[	]
		}

	s[*start]=1;
	g[*goal]=1;
		
	//	break;
		
}
int main()
 {
 int *i;
 int j;
 int *start_gpu,*goal_gpu;	
 int *start,*goal;	
 int *start_flag,*goal_flag;
 int *Dstart_flag,*Dgoal_flag;
 int *ROUTE;
 int *A; /* ホストメモリ用のポインタ*/
 int *DA;   /* デバイスメモリ用のポインタ*/
 long start_time,end_time,pre_time_start,pre_time_end,ret_time_start,ret_time_end;
 int *map;
 int *ii;
 int *end,*gend;
	
  ROUTE = (int *)malloc(sizeof(int)*MAPSIZE); /* 配列Aの領域確保*/
  goal_flag = (int *)malloc(sizeof(int)*MAPSIZE); /* 配列Aの領域確保*/
  i = (int *)malloc(sizeof(int)*MAPSIZE); /* 配列Aの領域確保*/
  start_flag = (int *)malloc(sizeof(int)*MAPSIZE); /* 配列Aの領域確保*/
  map = (int *)malloc(sizeof(int)*MAPSIZE); /* 配列Aの領域確保*/
  start = (int *)malloc(sizeof(int)); /* 配列Aの領域確保*/
  goal = (int *)malloc(sizeof(int)); /* 配列Aの領域確保*/
  end = (int *)malloc(sizeof(int)); /* 配列Aの領域確保*/
  *end=0;
  *start=WIDTH+2;
  *goal=MAPSIZE-WIDTH-3;
  start_flag[*start]=1;
  goal_flag[*goal]=1;
  	  for(*i=0;*i<MAPSIZE;*i=*i+1){
		if((*i/WIDTH)==0||(*i/WIDTH)==HEIGHT-1){
			ROUTE[*i]=WALL;
		}else if((*i%WIDTH)==0||(*i%WIDTH)==WIDTH-1){
				ROUTE[*i]=WALL;
		}else if((*i/WIDTH)%2==1){
				ROUTE[*i]=ROAD;
		}else if((*i/WIDTH)%2==0){
			if((*i%WIDTH)%2==0){
				ROUTE[*i]=ROAD;
			}else{
				ROUTE[*i]=WALL;
			}
		}
	}
	for(*i=0;*i<MAPSIZE;*i=*i+1){	
		printf("%04d,",ROUTE[*i]);
		if((*i%WIDTH)==(WIDTH-1)){
			printf("\n");
		}
	}
	printf("s%04d,g%d\n",*start,*goal);
 /* 配列A (GPU) の領域確保*/
 pre_time_start=clock();
 hipMalloc((int**)&Dstart_flag, sizeof(int)*MAPSIZE); 
 hipMalloc((int**)&Dgoal_flag, sizeof(int)*MAPSIZE); 
 hipMalloc((int**)&DA, sizeof(int)*MAPSIZE); 
 hipMalloc((int**)&A, sizeof(int)*MAPSIZE); 
 hipMalloc((int**)&start_gpu, sizeof(int)); 
 hipMalloc((int**)&goal_gpu, sizeof(int)); 
 hipMalloc((int**)&ii, sizeof(int)); 
 hipMalloc((int**)&gend, sizeof(int)); 
/* ホストメモリからデバイスメモリへコピー*/
 hipMemcpy(DA, map, sizeof(int)*MAPSIZE, hipMemcpyDefault);
 hipMemcpy( start_gpu,start, sizeof(int), hipMemcpyDefault);
 hipMemcpy( goal_gpu,goal, sizeof(int), hipMemcpyDefault);
 hipMemcpy( gend,end, sizeof(int), hipMemcpyDefault);
 pre_time_end=clock();
	printf("pre_time=%ld\n",pre_time_end-pre_time_start);
 /* GPUカーネル関数呼び出し*/
	start_time=clock();
	for(*i=0;;*i=*i+1){
 		hipMemcpy( ii,i, sizeof(int), hipMemcpyDefault);
 		hipMemcpy( A,ROUTE, sizeof(int)*MAPSIZE, hipMemcpyDefault);
 		hipMemcpy( Dstart_flag,start_flag, sizeof(int)*MAPSIZE, hipMemcpyDefault);
 		hipMemcpy( Dgoal_flag,goal_flag, sizeof(int)*MAPSIZE, hipMemcpyDefault);
 		hipMemcpy( gend,end ,sizeof(int), hipMemcpyDefault);
		bidirectional_search<<<(MAPSIZE+BS-1)/BS, BS>>>(ii,A,Dstart_flag,Dgoal_flag,start_gpu,goal_gpu,i,gend);
 		hipMemcpy( start_flag,Dstart_flag, sizeof(int)*MAPSIZE, hipMemcpyDefault);
 		hipMemcpy( goal_flag,Dgoal_flag, sizeof(int)*MAPSIZE, hipMemcpyDefault);
 		hipMemcpy( end,gend ,sizeof(int), hipMemcpyDefault);
 		hipMemcpy( ROUTE,A, sizeof(int)*MAPSIZE, hipMemcpyDefault);
//	printf("\n-----------------------ends\n");
		if(*end==1){
			if(j>2){
					break;
			}
			j++;
			
		}
	}
//	hipStreamSynchronize(stream);
	end_time=clock();
		
	printf("exe_time=%ld\n,",(end_time-start_time));
	for(*i=0;*i<MAPSIZE;*i=*i+1){	
		printf("%4d,",ROUTE[*i]);
		if((*i%WIDTH)==(WIDTH-1)){
			printf("\n");
		}
	}
 ret_time_start=clock();
	hipFree(DA);
	hipFree(A);
	hipFree(start_gpu);
	hipFree(ii);
	hipFree(goal_gpu);
	free(ROUTE);
	free(i);
	free(map);
	return;
 }
